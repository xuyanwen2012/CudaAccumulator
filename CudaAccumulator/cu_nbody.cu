#include "hip/hip_runtime.h"
#include "cu_nbody.cuh"
#include ""

__global__ void body_force(const int* a, const int* b, int* c, const int n)
{
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid < n)
	{
		c[tid] = a[tid] + b[tid];
	}
}

void compute_with_cuda()
{
	HANDLE_ERROR(hipSetDevice(0));

	constexpr int n_bodies = 1024;


	int a[n_bodies];
	int b[n_bodies];
	int c[n_bodies];

	for (int i = 0; i < n_bodies; ++i)
	{
		a[i] = i;
		b[i] = n_bodies * 10 + i;
	}

	int* dev_a;
	int* dev_b;
	int* dev_c;

	constexpr int bytes = n_bodies * sizeof(int);
	HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&dev_a), bytes));
	HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&dev_b), bytes));
	HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&dev_c), bytes));

	HANDLE_ERROR(hipMemcpy(dev_a, a, bytes, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, bytes, hipMemcpyHostToDevice));


	constexpr int block_size = 256;
	constexpr int n_blocks = (n_bodies + block_size - 1) / block_size;

	body_force <<<n_blocks, block_size>>>(dev_a, dev_b, dev_c, n_bodies);

	HANDLE_ERROR(hipMemcpy(c, dev_c, bytes, hipMemcpyDeviceToHost));

	for (int i = 0; i < 10; ++i)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	HANDLE_ERROR(hipDeviceReset());
}
