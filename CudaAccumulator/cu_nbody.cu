#include "hip/hip_runtime.h"
#include "cu_nbody.cuh"

__global__ void add(int a, int b, int* c)
{
	*c = a + b;
}

void compute_with_cuda()
{
	HANDLE_ERROR(hipSetDevice(0));

	int c;
	int* dev_c;
	HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&dev_c), sizeof(int)));

	add<<<1,1>>>(2, 7, dev_c);

	HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

	printf("2+7=%d\n", c);
	hipFree(dev_c);

	HANDLE_ERROR(hipDeviceReset());
}
